#include "hip/hip_runtime.h"
#include "bellmanFordCompleteGraphGpuParallel.h"
#include "bellmanFordCompleteGraphSequential.h"

// TODO: Use better values.
#define INFINIT_DISTANCE 1000000
#define NO_PREV 100000
#define DEBUG 1

static inline void initArrays(float *distanceArray, long size) {
    for (unsigned long i = 0; i < size; i++) {
        distanceArray[i] = INFINIT_DISTANCE;
    }
}

static void fillGpuGraphRandom(GpuGraph *graph) {
    if (!graph) {
        return;
    }
    srand48(10);
    for (unsigned long i = 0; i < graph->size * graph->size; i++) {
        graph->adjMatrix1D[i] = drand48();
    }
}

static CompleteGraph buildRandomCompleteGraph(unsigned int size) {
    CompleteGraph graph = createCompleteGraph(size);
    if (graph.error) {
        return graph;
    }

    unsigned int y, x;

    srand48(10);
    for (y = 0; y < size; y++) {
        for (x = 0; x < size; x++) {
            graph.adjMatrix[y][x] = (float) drand48();
            if(y == 0 && x == 0){
            }
        }
    }

    return graph;
}

static bool cmpDistArr(float* dist1, float* dist2, unsigned int size){
    if(!dist1 || !dist2){
        return false;
    }

    for(int i = 0; i < size; i++){
        if(dist1[i] != dist2[i]){
            return false;
        }
    }

    return true;
}

GpuGraph createGpuGraph(unsigned int size) {
    if (size > MAX_GRAPH_SIZE) {
        size = MAX_GRAPH_SIZE;
    }
    GpuGraph GpuGraph = {.size = size, .isDirected = false, .error = false};

    GpuGraph.dist = (float *) malloc(sizeof(float) * size);
    GpuGraph.adjMatrix1D = (float *) malloc(sizeof(float) * size * size);

    if (!GpuGraph.dist || !GpuGraph.adjMatrix1D) {
        exit(-101);
    }

    unsigned int i;

    for (i = 0; i < size * size; i++) {
        GpuGraph.adjMatrix1D[i] = 0;
    }
    return GpuGraph;
}


void destroyGpuGraph(GpuGraph *GpuGraph) {
    free(GpuGraph->dist);
    free(GpuGraph->adjMatrix1D);
}


__global__ void innerBellmanFord(float *adjMatrix1D, float *dist, unsigned int size, int *finished) {
    unsigned int x, y, currentMatrixPosition;
    currentMatrixPosition = threadIdx.x + blockIdx.x * blockDim.x;
    do {
        y = currentMatrixPosition / size;
        x = currentMatrixPosition & size;
        float weight = adjMatrix1D[currentMatrixPosition];
        if (dist[y] + weight < dist[x]) {
            dist[x] = dist[y] + weight;
            finished = 0;

        }
        currentMatrixPosition += gridDim.x * blockDim.x;
    } while (currentMatrixPosition < size * size);

}

double bellmanFordGpu(GpuGraph *graph, unsigned int startVertex, unsigned int blockSize, unsigned int threadNum) {

    // CPU Setup
    if (!graph || !graph->adjMatrix1D || !graph->dist) {
        return -1;
    }
    if(DEBUG) printf("Init arrays...\n");
    initArrays(graph->dist, graph->size);
    graph->dist[startVertex] = 0;
    double starttime, endtime;
    int *finished = (int*) malloc(sizeof(int));
    int *finishedGpu;
    unsigned int n, y, x, i;
    float *gpuadjMatrix1D;
    float *gpuDistArray;

    // GPU Setup
    if(DEBUG) printf("CUDA malloc...\n");
    CHECK(hipMalloc((void **) &gpuadjMatrix1D, sizeof(float) * graph->size * graph->size));
    CHECK(hipMalloc((void **) &gpuDistArray, sizeof(float) * graph->size));
    CHECK(hipMalloc((void **) &finishedGpu, sizeof(int)));
    if(DEBUG) printf("CUDA malloc done...\n");
    int grid = (graph->size * graph->size) / threadNum;

    double time = seconds();
    for (n = 0; n < graph->size; n++) {
        *finished = 1;
        if(DEBUG) printf("CUDA memcpy for n=%d...\n",n);
        CHECK(hipMemcpy(gpuadjMatrix1D, graph->adjMatrix1D, sizeof(float) * graph->size * graph->size,
                         hipMemcpyHostToDevice));
        CHECK(hipMemcpy(gpuDistArray, graph->dist, sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(finishedGpu, finished, sizeof(int), hipMemcpyHostToDevice));

        if(DEBUG) printf("Inner Bellmanford...\n");
        innerBellmanFord <<<grid, blockSize>>> (gpuadjMatrix1D, gpuDistArray, graph->size, finishedGpu);
        CHECK(hipDeviceSynchronize());

        if(DEBUG) printf("CUDA memcpy back...\n");
        CHECK(hipMemcpy(graph->adjMatrix1D, gpuadjMatrix1D, sizeof(float) * graph->size * graph->size,
                         hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(graph->dist, gpuDistArray, sizeof(int), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(finished, finishedGpu, sizeof(int), hipMemcpyDeviceToHost));

        CHECK(hipGetLastError());

        if (*finished) {
            break;
        }
    }
    if(DEBUG) printf("Done...\n");
    time = seconds() - time;

    CHECK(hipFree(gpuadjMatrix1D));
    CHECK(hipFree(gpuDistArray));
    CHECK(hipFree(finishedGpu));

    CHECK(hipDeviceReset());

    return time;
}

int main() {
    if(DEBUG) printf("Starting GPU Test...\n");

    // init locals
    int dev = 0;
    unsigned int n = 10000;
    unsigned int blockSize, threadsPerBlock;
    if(DEBUG) printf("Create graph...\n");
    GpuGraph graph = createGpuGraph(n);

    if(DEBUG) printf("Fill graph...\n");
    fillGpuGraphRandom(&graph);
    if(DEBUG) printf("Fill done...\n");
    CHECK(hipSetDevice(dev));
    blockSize = 512;
    threadsPerBlock = 512;
    if(DEBUG) printf("Run gpu bellman ford...\n");
    double time = bellmanFordGpu(&graph, 0, blockSize, threadsPerBlock);
    printf("result=%lf\n",time);

    CompleteGraph cpuGraph = buildRandomCompleteGraph(n);
    bellmanFord(&cpuGraph,0);
    bool check = cmpDistArr(cpuGraph.dist,graph.dist,graph.size);
    printf("check=%d\n",check);
    

}