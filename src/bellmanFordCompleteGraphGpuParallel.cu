#include "hip/hip_runtime.h"
#include "bellmanFordCompleteGraphGpuParallel.h"

// TODO: Use better values.
#define INFINIT_DISTANCE 1000000
#define NO_PREV 100000
#define DEBUG 1


// REGION: SEQU Graph
typedef struct CompleteGraph {
    unsigned int size; //< the number of vertices.
    bool isDirected; //< indicates if the graph is directed.
    bool error; //< a flag which will be true if any function call on the graph struct causes an error.
    float **adjMatrix; //< a 2D matrix with the dimensions of size * size, where every colume indicates the distance between 2 vertices.
    float *dist; //< Stores the distance to a start vertex. Can be filled with shortest path algorithm.
} CompleteGraph;

inline void initArraysSequ(float *distanceArray,long size) {
    unsigned long i;
    for (i = 0; i < size; i++) {
        distanceArray[i] = INFINIT_DISTANCE;
    }
}

void destroyCompleteGraph(CompleteGraph *completeGraph) {
    free(completeGraph->dist);
    unsigned int i;
    for (i = 0; i < completeGraph->size; i++) {
        if (completeGraph->adjMatrix[i]) {
            free(completeGraph->adjMatrix[i]);
        }
    }
    free(completeGraph->adjMatrix);
}

CompleteGraph createCompleteGraph(unsigned int size) {
    if (size > MAX_GRAPH_SIZE) {
        size = MAX_GRAPH_SIZE;
    }
    CompleteGraph completeGraph = {.size = size, .isDirected = false};

    completeGraph.dist = (float *) malloc(sizeof(float) * size);
    completeGraph.adjMatrix = (float **) malloc(sizeof(float *) * size);

    if (!completeGraph.dist || !completeGraph.adjMatrix) {
        destroyCompleteGraph(&completeGraph);
        return {};
    }

    unsigned int i, x;

    for (i = 0; i < size; i++) {
        completeGraph.adjMatrix[i] = (float *) malloc(sizeof(float) * size);
        if (!completeGraph.adjMatrix[i]) {
            destroyCompleteGraph(&completeGraph);
            return {};
        }
        if (i == 0) {
            for (x = 0; x < size; x++) {
                completeGraph.adjMatrix[i][x] = 0;
            }
        } else {
            memcpy(completeGraph.adjMatrix[i], completeGraph.adjMatrix[0], sizeof(float) * size);
        }

    }
    return completeGraph;
}

double bellmanFord(CompleteGraph *graph, unsigned int startVertex) {
    if (!graph || !graph->adjMatrix || !graph->dist) {
        return -1;
    }
    initArraysSequ(graph->dist, graph->size);
    graph->dist[startVertex] = 0;
    double startTime, endTime;
    bool finished;
    unsigned int n, y, x;
    startTime = seconds();
    for (n = 0; n < graph->size; n++) {
        finished = true;
        for (y = 0; y < graph->size; y++) {
            for (x = 0; x < graph->size; x++) {
                float weight = graph->adjMatrix[y][x];
                if (graph->dist[y] + weight < graph->dist[x]) {
                    graph->dist[x] = graph->dist[y] + weight;
                    finished = false;
                }
            }
        }
        if (finished) {
            break;
        }
    }
    endTime = seconds();
    return endTime - startTime;
}

static inline void initArrays(float *distanceArray, long size) {
    for (unsigned long i = 0; i < size; i++) {
        distanceArray[i] = INFINIT_DISTANCE;
    }
}

static void fillGpuGraphRandom(GpuGraph *graph) {
    if (!graph) {
        return;
    }
    srand48(10);
    for (unsigned long i = 0; i < graph->size * graph->size; i++) {
        graph->adjMatrix1D[i] = drand48();
    }
}

static CompleteGraph buildRandomCompleteGraph(unsigned int size) {
    CompleteGraph graph = createCompleteGraph(size);
    if (graph.error) {
        return graph;
    }

    unsigned int y, x;

    srand48(10);
    for (y = 0; y < size; y++) {
        for (x = 0; x < size; x++) {
            graph.adjMatrix[y][x] = (float) drand48();
            if(y == 0 && x == 0){
            }
        }
    }

    return graph;
}

static bool cmpDistArr(CompleteGraph* completeGraph, GpuGraph* gpuGraph, unsigned int size){
    if(!gpuGraph->dist || !completeGraph->dist){
        if(DEBUG) printf("Diff error 1\n");
        return false;
    }
    int i,y;
    if(DEBUG){
        for(i = 0; i < size; i++){
            for(y = 0; y < size; y++){
                if(completeGraph->adjMatrix[i][y] != gpuGraph->adjMatrix1D[y+(i*size)]){
                    if(DEBUG) printf("Diff error 2 for i=%d & y=%d\n",i,y);
                    return false;
                }
            }
        }
    }
    for(i = 0; i < size; i++){
        if(gpuGraph->dist[i] != completeGraph->dist[i]){
            if(DEBUG) printf("Diff error 3 for i=%d\n",i);
            if(DEBUG) printf("GPU: %lf vs CPU:%lf\n",gpuGraph->dist[i], completeGraph->dist[i]);
            return false;
        }
    }

    return true;
}

GpuGraph createGpuGraph(unsigned int size) {
    if (size > MAX_GRAPH_SIZE) {
        size = MAX_GRAPH_SIZE;
    }
    GpuGraph GpuGraph = {.size = size, .isDirected = false, .error = false};

    GpuGraph.dist = (float *) malloc(sizeof(float) * size);
    GpuGraph.adjMatrix1D = (float *) malloc(sizeof(float) * size * size);

    if (!GpuGraph.dist || !GpuGraph.adjMatrix1D) {
        exit(-101);
    }

    unsigned int i;

    for (i = 0; i < size * size; i++) {
        GpuGraph.adjMatrix1D[i] = 0;
    }
    return GpuGraph;
}

void destroyGpuGraph(GpuGraph *GpuGraph) {
    free(GpuGraph->dist);
    free(GpuGraph->adjMatrix1D);
}

__global__ void innerBellmanFord(float *adjMatrix1D, float *dist, unsigned int size, int *finished) {
    unsigned int x, y, currentMatrixPosition;
    currentMatrixPosition = threadIdx.x + blockIdx.x * blockDim.x;
    do {
        x = currentMatrixPosition / size;
        y = currentMatrixPosition & size;
        float weight = adjMatrix1D[currentMatrixPosition];
        if (dist[y] + weight < dist[x]) {
            dist[x] = dist[y] + weight;
            *finished = 0;

        }
        currentMatrixPosition += gridDim.x * blockDim.x;
    } while (currentMatrixPosition < size * size);

}

double bellmanFordGpu(GpuGraph *graph, unsigned int startVertex, unsigned int blockSize, unsigned int threadNum) {

    if (!graph || !graph->adjMatrix1D || !graph->dist) {
        return -1;
    }
    if(DEBUG) printf("Init arrays...\n");
    initArrays(graph->dist, graph->size);
    graph->dist[startVertex] = 0;
    int *finished = (int*) malloc(sizeof(int));
    int *finishedGpu;
    unsigned int n;
    float *gpuadjMatrix1D;
    float *gpuDistArray;
    unsigned long size2D = sizeof(float) * graph->size * graph->size;

    // GPU Setup
    if(DEBUG) printf("CUDA malloc...\n");
    CHECK(hipMalloc((void **) &gpuadjMatrix1D, size2D));
    CHECK(hipMalloc((void **) &gpuDistArray, sizeof(float) * graph->size));
    CHECK(hipMalloc((void **) &finishedGpu, sizeof(int)));
    if(DEBUG) printf("CUDA malloc done...\n");
    int grid = (graph->size * graph->size) / threadNum;

    double time = seconds();
    for (n = 0; n < graph->size; n++) {
        *finished = 1;
        if(DEBUG) printf("CUDA memcpy for n=%d...\n",n);
        CHECK(hipMemcpy(gpuadjMatrix1D, graph->adjMatrix1D, size2D, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(gpuDistArray, graph->dist, sizeof(float) * graph->size, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(finishedGpu, finished, sizeof(int), hipMemcpyHostToDevice));

        if(DEBUG) printf("Inner Bellmanford...\n");
        innerBellmanFord <<<grid, blockSize>>> (gpuadjMatrix1D, gpuDistArray, graph->size, finishedGpu);
        CHECK(hipDeviceSynchronize());

        if(DEBUG) printf("CUDA memcpy back...\n");
        CHECK(hipMemcpy(graph->adjMatrix1D, gpuadjMatrix1D, size2D, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(graph->dist, gpuDistArray, sizeof(float) * graph->size, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(finished, finishedGpu, sizeof(int), hipMemcpyDeviceToHost));

        CHECK(hipGetLastError());

        if (*finished) {
            printf("True Finished with n=%d...\n",n);
            break;
        }
    }
    if(DEBUG) printf("Done...\n");
    time = seconds() - time;

    CHECK(hipFree(gpuadjMatrix1D));
    CHECK(hipFree(gpuDistArray));
    CHECK(hipFree(finishedGpu));

    CHECK(hipDeviceReset());

    free(finished);

    return time;
}

int main() {
    if(DEBUG) printf("Starting GPU Test...\n");

    // init locals
    int dev = 0;
    unsigned int n = 10000;
    unsigned int blockSize, threadsPerBlock;
    if(DEBUG) printf("Create graph...\n");
    GpuGraph graph = createGpuGraph(n);

    if(DEBUG) printf("Fill graph...\n");
    fillGpuGraphRandom(&graph);
    if(DEBUG) printf("Fill done...\n");
    CHECK(hipSetDevice(dev));
    blockSize = 512;
    threadsPerBlock = 512;
    if(DEBUG) printf("Run gpu bellman ford...\n");
    double time = bellmanFordGpu(&graph, 0, blockSize, threadsPerBlock);
    printf("result=%lf\n",time);
    if(DEBUG) printf("Build cpu graph...\n");
    CompleteGraph cpuGraph = buildRandomCompleteGraph(n);
    if(DEBUG) printf("Run cpu bellman-ford...\n");
    bellmanFord(&cpuGraph,0);
    if(DEBUG) printf("Run check...\n");
    bool check = cmpDistArr(&cpuGraph,&graph,graph.size);
    printf("check=%d\n",check);
    

}